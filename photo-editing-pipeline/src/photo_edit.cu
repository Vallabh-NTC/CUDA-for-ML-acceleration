#include "hip/hip_runtime.h"
// photo_edit.cu
// Re-process the selected photo whenever editor.json changes.
// Saves to "<gallery>/<basename>-edited.jpg" (overwritten each update).

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <string>
#include <vector>
#include <sys/stat.h>
#include <unistd.h>
#include <fcntl.h>
#include <cerrno>
#include <cstring>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// ---------- tiny helpers ----------
static inline bool file_exists(const std::string& p) {
    struct stat st; return ::stat(p.c_str(), &st) == 0;
}
static inline time_t file_mtime(const std::string& p) {
    struct stat st; if (::stat(p.c_str(), &st) != 0) return 0; return st.st_mtime;
}
static inline std::string basename_no_ext(const std::string& path) {
    size_t s = path.find_last_of("/\\"); std::string name = (s==std::string::npos)? path : path.substr(s+1);
    size_t dot = name.find_last_of('.'); if (dot==std::string::npos) return name; return name.substr(0,dot);
}
static inline std::string dir_of(const std::string& path) {
    size_t s = path.find_last_of("/\\"); return (s==std::string::npos)? std::string(".") : path.substr(0,s);
}

// -------- params (subset aligned with your editor.json keys) ----------
struct Params {
    bool   enable      = true;
    float  contrast    = 1.00f;  // 0.50 .. 1.80
    float  brightness  = 0.00f;  // -1.00 .. +1.00 (adds after [0..1] mapping)
    float  gamma       = 1.00f;  // 0.50 .. 2.00
    float  saturation  = 1.00f;  // 0.00 .. 4.00
};

// very tolerant JSON puller (like your ColorConfigIO)
static float pullFloat(const std::string& s, const char* key, float def) {
    std::string pat = std::string("\"") + key + "\"";
    size_t k = s.find(pat); if (k==std::string::npos) return def;
    k = s.find(':', k); if (k==std::string::npos) return def;
    size_t e = s.find_first_of(",}\n\r", k+1);
    std::string num = s.substr(k+1, (e==std::string::npos?s.size():e)-(k+1));
    try { return std::stof(num); } catch(...) { return def; }
}
static bool pullBool(const std::string& s, const char* key, bool def) {
    std::string pat = std::string("\"") + key + "\"";
    size_t k = s.find(pat); if (k==std::string::npos) return def;
    k = s.find(':', k); if (k==std::string::npos) return def;
    std::string val = s.substr(k+1, 8);
    return val.find("true") != std::string::npos;
}

static Params read_params_from_editor_json(const std::string& jsonPath, const Params& prev) {
    FILE* f = ::fopen(jsonPath.c_str(), "rb");
    if (!f) return prev;
    std::string data;
    char buf[4096];
    size_t n;
    while ((n = std::fread(buf,1,sizeof(buf),f))>0) data.append(buf,n);
    std::fclose(f);

    Params p = prev;
    p.enable     = pullBool (data, "enable",     p.enable);
    p.contrast   = pullFloat(data, "contrast",   p.contrast);
    p.brightness = pullFloat(data, "brightness", p.brightness);
    p.gamma      = pullFloat(data, "gamma",      p.gamma);
    p.saturation = pullFloat(data, "saturation", p.saturation);

    // clamp to your runtime ranges
    auto clamp = [](float v, float lo, float hi){ return v<lo?lo:(v>hi?hi:v); };
    p.contrast   = clamp(p.contrast,   0.50f, 1.80f);
    p.brightness = clamp(p.brightness, -1.0f, 1.0f);
    p.gamma      = clamp(p.gamma,       0.5f, 2.0f);
    p.saturation = clamp(p.saturation,  0.0f, 4.0f);

    return p;
}

// -------- CUDA bits (simple RGB kernel) --------
__device__ inline float clampf(float v, float a, float b){ return v<a?a:(v>b?b:v); }
__device__ inline float3 rgb2hsv(float3 c) {
    float mx = fmaxf(c.x, fmaxf(c.y, c.z));
    float mn = fminf(c.x, fminf(c.y, c.z));
    float d  = mx - mn;
    float h = 0.f;
    if (d > 1e-6f) {
        if (mx == c.x)      h = fmodf(((c.y - c.z) / d), 6.f);
        else if (mx == c.y) h = ((c.z - c.x) / d) + 2.f;
        else                h = ((c.x - c.y) / d) + 4.f;
        h *= 60.f; if (h < 0.f) h += 360.f;
    }
    float s = (mx <= 0.f) ? 0.f : (d / mx);
    float v = mx;
    return make_float3(h,s,v);
}
__device__ inline float3 hsv2rgb(float3 h) {
    float H=h.x, S=h.y, V=h.z;
    if (S<=1e-6f) return make_float3(V,V,V);
    float C = V*S;
    float X = C*(1.f - fabsf(fmodf(H/60.f,2.f)-1.f));
    float m = V - C;
    float3 r;
    if      (H<60)   r=make_float3(C,X,0);
    else if (H<120)  r=make_float3(X,C,0);
    else if (H<180)  r=make_float3(0,C,X);
    else if (H<240)  r=make_float3(0,X,C);
    else if (H<300)  r=make_float3(X,0,C);
    else             r=make_float3(C,0,X);
    r.x += m; r.y += m; r.z += m;
    return r;
}

__global__ void apply_params_rgb(
    const unsigned char* __restrict__ in,
    unsigned char* __restrict__ out,
    int W, int H, int stride,
    float contrast, float brightness, float gamma, float saturation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x>=W || y>=H) return;

    const unsigned char* src = in + y*stride + 3*x;
    unsigned char*       dst = out + y*stride + 3*x;

    float r = src[0]/255.f, g = src[1]/255.f, b = src[2]/255.f;

    // contrast around mid-gray 0.5
    r = (r - 0.5f) * contrast + 0.5f;
    g = (g - 0.5f) * contrast + 0.5f;
    b = (b - 0.5f) * contrast + 0.5f;

    // global gamma
    float invg = (gamma>1e-6f) ? (1.0f/gamma) : 1.f;
    r = powf(clampf(r,0,1), invg);
    g = powf(clampf(g,0,1), invg);
    b = powf(clampf(b,0,1), invg);

    // brightness (full-range offset)
    r = clampf(r + brightness, 0.f, 1.f);
    g = clampf(g + brightness, 0.f, 1.f);
    b = clampf(b + brightness, 0.f, 1.f);

    // saturation in HSV
    float3 hsv = rgb2hsv(make_float3(r,g,b));
    hsv.y *= saturation;
    hsv.y = clampf(hsv.y, 0.f, 4.f);
    float3 rr = hsv2rgb(hsv);

    dst[0] = (unsigned char)(clampf(rr.x,0.f,1.f)*255.f + 0.5f);
    dst[1] = (unsigned char)(clampf(rr.y,0.f,1.f)*255.f + 0.5f);
    dst[2] = (unsigned char)(clampf(rr.z,0.f,1.f)*255.f + 0.5f);
}

// -------- main loop --------
int main(int argc, char** argv) {
    std::string inPath, galleryDir, editorJson = "/home/moviemaker/editor.json";

    // args we already pass from jetson_editor
    for (int i=1;i<argc;i++) {
        std::string a = argv[i];
        auto need = [&](const char* flag){
            if (a==flag && i+1<argc){ return true; } return false;
        };
        if (need("--input"))       { inPath = argv[++i]; continue; }
        if (need("--gallery"))     { galleryDir = argv[++i]; continue; }
        if (need("--editor-json")) { editorJson = argv[++i]; continue; }
        if (a=="--help"||a=="-h"){
            std::fprintf(stderr,
                "Usage: %s --input <img> --gallery <dir> [--editor-json <path>]\n", argv[0]);
            return 1;
        }
    }

    if (inPath.empty()) {
        std::fprintf(stderr, "[PHOTO] ERROR: --input is required\n");
        return 2;
    }
    if (galleryDir.empty()) galleryDir = dir_of(inPath);

    const std::string base   = basename_no_ext(inPath);
    const std::string outJpg = galleryDir + "/" + base + "-edited.jpg";

    std::printf("[PHOTO] Photo editing started\n");
    std::printf("[PHOTO] Selected file: %s\n", base.c_str());
    std::printf("[PHOTO] Full path:     %s\n", inPath.c_str());
    std::printf("[PHOTO] Output (fixed): %s\n", outJpg.c_str());
    std::printf("[PHOTO] Watching editor.json: %s\n", editorJson.c_str());

    // Load original once (RGB8)
    int W=0,H=0,N=0;
    stbi_uc* img = stbi_load(inPath.c_str(), &W, &H, &N, 3);
    if (!img) {
        std::fprintf(stderr, "[PHOTO] ERROR: failed to load image: %s\n", inPath.c_str());
        return 3;
    }
    const size_t stride = (size_t)W * 3;
    const size_t bytes  = (size_t)H * stride;

    // device buffers (keep alive)
    unsigned char *d_in=nullptr, *d_out=nullptr;
    hipMalloc(&d_in,  bytes);
    hipMalloc(&d_out, bytes);
    hipMemcpy(d_in, img, bytes, hipMemcpyHostToDevice);

    // defaults (will be overwritten by json on first pass if present)
    Params params;
    time_t last_mtime = 0;

    auto run_once = [&](const Params& p){
        dim3 b(16,16), g((W+b.x-1)/b.x,(H+b.y-1)/b.y);
        apply_params_rgb<<<g,b>>>(d_in, d_out, W, H, (int)stride,
                                  p.contrast, p.brightness, p.gamma, p.saturation);
        hipDeviceSynchronize();

        std::vector<unsigned char> host(bytes);
        hipMemcpy(host.data(), d_out, bytes, hipMemcpyDeviceToHost);

        if (!stbi_write_jpg(outJpg.c_str(), W, H, 3, host.data(), 90)) {
            std::fprintf(stderr, "[PHOTO] ERROR: write failed: %s\n", outJpg.c_str());
        } else {
            std::printf("[PHOTO] wrote: %s  (contrast=%.3f, bright=%.3f, gamma=%.3f, sat=%.3f)\n",
                        outJpg.c_str(), p.contrast, p.brightness, p.gamma, p.saturation);
            // fsync to make browser see it immediately
            int fd = ::open(outJpg.c_str(), O_RDONLY);
            if (fd>=0) { ::fsync(fd); ::close(fd); }
        }
    };

    // force an initial run (even if editor.json missing)
    run_once(params);

    // watch loop
    while (true) {
        time_t mt = file_mtime(editorJson);
        if (mt != 0 && mt != last_mtime) {
            last_mtime = mt;
            Params p2 = read_params_from_editor_json(editorJson, params);
            params = p2;
            if (params.enable) {
                run_once(params);
            } else {
                std::printf("[PHOTO] enable=false -> skipping write\n");
            }
        }
        // keep it light; your parent process will kill us on "stop"
        usleep(150 * 1000); // 150 ms
    }

    // never reached in normal flow
    hipFree(d_in); hipFree(d_out);
    stbi_image_free(img);
    return 0;
}
