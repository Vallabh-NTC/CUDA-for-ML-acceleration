#include "hip/hip_runtime.h"
#include "wire_lineedge.cuh"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>

namespace {

// clamp helpers
__device__ __forceinline__ int clampi(int v,int a,int b){
    return v<a ? a : (v>b ? b : v);
}
__device__ __forceinline__ float clampf(float v,float a,float b){
    return v<a ? a : (v>b ? b : v);
}

// ============== helpers: pitched bilinear (Y) / nearest (UV) ==============

__device__ __forceinline__ float bilinear_Y(
    const uint8_t* Y, int pY, int W, int H,
    float xf, float yf)
{
    xf = clampf(xf, 0.0f, (float)(W-1));
    yf = clampf(yf, 0.0f, (float)(H-1));
    int x0 = (int)floorf(xf), y0 = (int)floorf(yf);
    int x1 = min(W-1, x0+1), y1 = min(H-1, y0+1);
    float ax = xf - x0, ay = yf - y0;

    int i00 = Y[y0 * pY + x0];
    int i10 = Y[y0 * pY + x1];
    int i01 = Y[y1 * pY + x0];
    int i11 = Y[y1 * pY + x1];

    float v0 = i00*(1.0f-ax) + i10*ax;
    float v1 = i01*(1.0f-ax) + i11*ax;
    return v0*(1.0f-ay) + v1*ay;
}

__device__ __forceinline__ void fetch_UV_nearest(
    const uint8_t* UV, int pUV, int W, int H,
    float xf, float yf, uint8_t& U, uint8_t& V)
{
    // NV12 chroma half-res: pick nearest chroma sample to (xf, yf)
    float cxf = 0.5f * xf;
    float cyf = 0.5f * yf;
    int CW = (W + 1) >> 1, CH = (H + 1) >> 1;
    int cu = clampi((int)floorf(cxf + 0.5f), 0, CW-1);
    int cv = clampi((int)floorf(cyf + 0.5f), 0, CH-1);
    const uint8_t* p = UV + cv * pUV + cu * 2;
    U = p[0];
    V = p[1];
}

} // anon

namespace wire {

// =================== simplest constant-shift inpaint per mask ==============
//
// For each pixel (x,y) where mask!=0:
//   Y(x,y)  = Y(x+dx, y+dy)  [bilinear]
//   UV(x,y) = UV(x+dx, y+dy) [nearest]
// ===========================================================================

__global__ void k_apply_mask_shift_Y(
    uint8_t* __restrict__ Y, int pY,
    const uint8_t* __restrict__ mask, int pMask,
    int W, int H, float dx, float dy)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= W || y >= H) return;
    if (mask[y * pMask + x] == 0) return;

    float xs = (float)x + dx;
    float ys = (float)y + dy;

    float Ydon = bilinear_Y(Y, pY, W, H, xs, ys);
    int val = (int)(Ydon + 0.5f);
    val = val < 0 ? 0 : (val > 255 ? 255 : val);
    Y[y * pY + x] = (uint8_t)val;
}

__global__ void k_apply_mask_shift_UV(
    uint8_t* __restrict__ UV, int pUV,
    const uint8_t* __restrict__ mask, int pMask,
    int W, int H, float dx, float dy)
{
    int cu = blockIdx.x * blockDim.x + threadIdx.x;
    int cv = blockIdx.y * blockDim.y + threadIdx.y;
    int CW = (W + 1) >> 1, CH = (H + 1) >> 1;
    if (cu >= CW || cv >= CH) return;

    // if ANY of the covered luma pixels are masked, write UV
    int x0 = cu * 2, y0 = cv * 2;
    uint8_t any = 0;
    if (y0 < H) {
        if (x0   < W) any |= mask[y0 * pMask + x0];
        if (x0+1 < W) any |= mask[y0 * pMask + x0+1];
    }
    if (y0+1 < H) {
        if (x0   < W) any |= mask[(y0+1) * pMask + x0];
        if (x0+1 < W) any |= mask[(y0+1) * pMask + x0+1];
    }
    if (!any) return;

    // donor at shifted location (map to chroma grid internally)
    float x_center = (float)cu * 2.0f + 0.5f;
    float y_center = (float)cv * 2.0f + 0.5f;
    float xs = x_center + dx;
    float ys = y_center + dy;

    uint8_t U, V;
    fetch_UV_nearest(UV, pUV, W, H, xs, ys, U, V);

    uint8_t* p = UV + cv * pUV + cu * 2;
    p[0] = U; p[1] = V;
}

void apply_mask_shift_nv12(
    uint8_t* dY,  int pitchY,
    uint8_t* dUV, int pitchUV,
    int W, int H,
    const uint8_t* dMask, int maskPitch,
    float dx, float dy,
    hipStream_t stream)
{
    dim3 blk(32,8), grd((W+31)/32, (H+7)/8);
    k_apply_mask_shift_Y <<<grd, blk, 0, stream>>> (dY, pitchY, dMask, maskPitch, W, H, dx, dy);

    int CW=(W+1)>>1, CH=(H+1)>>1;
    dim3 blkUV(32,8), grdUV((CW+31)/32,(CH+7)/8);
    k_apply_mask_shift_UV<<<grdUV, blkUV, 0, stream>>> (dUV, pitchUV, dMask, maskPitch, W, H, dx, dy);
}

} // namespace wire
