// image-correction-pipeline/src/nvivafilter_rectify.cu
// -----------------------------------------------------------------------------
// This file is the "glue" between GStreamer’s gst-nvivafilter and CUDA
// rectification kernel. It maps the incoming NVMM EGLImage to a CUDA-accessible
// frame (CUeglFrame), checks that it is pitch-linear ABGR, and then calls the
// kernel launcher. Runtime tunables are pulled from a JSON file via the
// RuntimeControls helper.
//
// LIFECYCLE (gst-nvivafilter):
//   init()         → called once when the .so is loaded; we assign our callbacks
//   fPreProcess    → called when the pipeline goes to PLAYING; we allocate ctx
//   fGPUProcess    → called for each input EGLImage buffer; we run the kernel
//   fPostProcess   → called when the pipeline stops; we clean up
//
// THREADING:
//   - GStreamer calls fGPUProcess() on the streaming thread.GPU work is run on a CUDA stream 
//     and synchronized before returning, ensuring the frame is ready when passed downstream.
//   - RuntimeControls runs a separate, tiny thread to hot-reload JSON.
//
// FORMAT ASSUMPTION:
//   - We hard-require CU_EGL_COLOR_FORMAT_ABGR (i.e., RGBA pixels in ABGR enum).
//   - If your camera is UYVY/YUY2/NV12, add: nvvidconv ! video/x-raw(memory:NVMM),format=RGBA
//     *before* the nvivafilter in your pipeline.
//
// DEPENDENCY:
//   - "customer_functions.h" comes from Jetson Multimedia API (in targetfs).
// -----------------------------------------------------------------------------
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaEGL.h>

#include "customer_functions.h"      // from /usr/src/jetson_multimedia_api/samples/gst-nvivafilter
#include "runtime_controls.hpp"
#include "rectify_kernels.cuh"

#define CUCHK(x) do { hipError_t e=(x); if(e!=hipSuccess){ \
  const char* s=nullptr; hipDrvGetErrorString(e,&s); \
  fprintf(stderr,"CUDA-DRIVER error %d (%s) at %s:%d\n",(int)e,s?s:"?",__FILE__,__LINE__); }} while(0)
#define CUDACHK(x) do { hipError_t e=(x); if(e!=hipSuccess){ \
  fprintf(stderr,"CUDA-RUNTIME error %s at %s:%d\n", hipGetErrorString(e),__FILE__,__LINE__); }} while(0)

// Per-pipeline instance context. Kept minimal on purpose.
struct FilterCtx {
    RuntimeControls* controls = nullptr; // JSON-backed, hot-reloaded parameters
    hipStream_t stream = nullptr;       // CUDA stream used for processing
};

// Called by gst-nvivafilter when the pipeline transitions to PLAYING.
// We allocate our context, create a CUDA stream, and start the JSON watcher.
static void pre_process(void **, unsigned int*, unsigned int*, unsigned int*, unsigned int*,
                        ColorFormat*, unsigned int, void **usrptr)
{
    auto* ctx = new FilterCtx();
    CUDACHK(hipStreamCreate(&ctx->stream));

    // Controls file path:
    //  - default: /opt/rectify/controls.json
    //  - override via env: RECTIFY_CONTROLS_JSON=/path/to/controls.json
    const char* envp = std::getenv("RECTIFY_CONTROLS_JSON");
    std::string path = (envp && *envp) ? std::string(envp) : "/opt/rectify/controls.json";
    ctx->controls = new RuntimeControls(path);

    *usrptr = ctx;
    std::fprintf(stderr, "[nvivafilter_rectify] pre_process: watching %s\n", path.c_str());
}

// Called after streaming stops. We dispose the watcher and the CUDA stream.
static void post_process(void **, unsigned int*, unsigned int*, unsigned int*, unsigned int*,
                         ColorFormat*, unsigned int, void **usrptr)
{
    auto* ctx = (FilterCtx*)(*usrptr);
    if (!ctx) return;
    if (ctx->controls) delete ctx->controls;
    if (ctx->stream) hipStreamDestroy(ctx->stream);
    delete ctx;
    *usrptr = nullptr;
}

// Called for each input frame (EGLImage in NVMM). We map the EGLImage to a
// CUeglFrame, verify format, run the CUDA kernel in-place, then unmap.
static void gpu_process(EGLImageKHR image, void **usrptr)
{
    auto* ctx = (FilterCtx*)(*usrptr);
    if (!ctx) return;

    // Map EGLImage (NVMM) to CUDA via the driver API
    hipGraphicsResource_t res = nullptr;
    CUeglFrame frame{};
    CUCHK(cuGraphicsEGLRegisterImage(&res, image, CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE));
    CUCHK(cuGraphicsResourceGetMappedEglFrame(&frame, res, 0, 0));

    // We only handle pitch-linear ABGR (RGBA data layout recognized as ABGR enum)
    if (frame.frameType != CU_EGL_FRAME_TYPE_PITCH ||
        frame.eglColorFormat != CU_EGL_COLOR_FORMAT_ABGR) {
        std::fprintf(stderr, "[nvivafilter_rectify] Unsupported frame (type=%d col=%d). "
                             "Ensure nvvidconv -> RGBA before nvivafilter.\n",
                     frame.frameType, frame.eglColorFormat);
        CUCHK(hipGraphicsUnregisterResource(res));
        return;
    }

    // In-place processing: src == dst (safe because each output sample only reads
    // the snapshot at that position; if you ever add multi-tap filters that read
    // *neighboring* outputs, consider using a temporary copy to avoid hazards).
    uint8_t* d_ptr = (uint8_t*)frame.frame.pPitch[0];
    int pitch = frame.pitch;
    int w = frame.width;
    int h = frame.height;

    // Pull current parameters (single POD copy) and launch kernel
    RectifyConfig cfg = ctx->controls->get();
    launch_rectify_kernel(d_ptr, w, h, pitch, d_ptr, w, h, pitch, cfg, ctx->stream);

    // Synchronize to guarantee the GPU writes are complete before GStreamer
    // passes the buffer downstream.
    CUDACHK(hipStreamSynchronize(ctx->stream));

    // Unmap the resource so GStreamer can continue
    CUCHK(hipGraphicsUnregisterResource(res));
}

// Required by gst-nvivafilter to resolve our callbacks.
extern "C" void init(CustomerFunction* f) {
    f->fPreProcess  = pre_process;
    f->fGPUProcess  = gpu_process;
    f->fPostProcess = post_process;
}
extern "C" void deinit(void) {}
